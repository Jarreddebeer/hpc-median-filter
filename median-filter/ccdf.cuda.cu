#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define BLOCKSIZE 16

extern "C" {
#include "ccdf.h"
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__ int get_median_ccdf(
    int* sm,
    int sidx,
    int filt_rad,
    int sm_filt_len)
{
    int min = sm[sidx];
    int max = sm[sidx];
    // get min and max
    for (int i = -filt_rad; i <= filt_rad; i++)
    {
        for (int j = -filt_rad; j <= filt_rad; j++)
        {
            if (sm[sidx + i * sm_filt_len + j] < min)
                min = sm[sidx + i * sm_filt_len + j];
            else if (sm[sidx + i * sm_filt_len + j] > max)
                max = sm[sidx + i * sm_filt_len + j];
        }
    }
    // binary search on values for pivot
    int mid = (min + max) / 2;
    int count;
    while (min < max)
    {
        count = 0;
        for (int i = -filt_rad; i <= filt_rad; i++)
        {
            for (int j = -filt_rad; j <= filt_rad; j++)
            {
                if (sm[sidx + i * sm_filt_len + j] < mid) count++;
            }
        }
        if (count == (filt_rad / 2)) break;
        else if (count < (filt_rad / 2)) min = mid + 1;
        else max = mid - 1;
        mid = (min + max) / 2;
    }
    // get index of closest item in window
    int median = sm[sidx];
    int dist = abs(median - mid);
    for (int i = -filt_rad; i <= filt_rad; i++)
    {
        for (int j = -filt_rad; j <= filt_rad; j++)
        {
            int dist2 = abs(sm[sidx + i * sm_filt_len + j] - mid);
            if (dist2 < dist)
            {
                 median = sm[sidx + i * sm_filt_len + j];
                 dist = dist2;
            }
        }
    }
    // return median
    return median;
}

__global__ void median_filter_ccdf(
    int* d_in,
    int* d_out,
    int gridLen,
    int grid_filt_len,
    int filt_rad,
    int sm_filt_len)
{
    extern __shared__ int sm[];
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int sidx = sm_filt_len * (filt_rad + ty) + (filt_rad + tx);

    int x = blockDim.x * blockIdx.x + tx;
    int y = blockDim.y * blockIdx.y + ty;
    int idx = grid_filt_len * (filt_rad + y) + (filt_rad + x);

    __syncthreads();

    if (x < gridLen && y < gridLen)
    {

        // load pixels into shared memory, padded on host
        sm[sidx] = d_in[idx];

        // border pixels
        if (tx == 0)
        { // left border
            if (ty == 0)
            { // top corner
                for (int i = 1; i <= filt_rad; i++) for (int j = 1; j <= filt_rad; j++)
                {
                    sm[sidx - i - j * sm_filt_len] = d_in[idx - i - j * grid_filt_len];
                }
            }
            else if (ty == blockDim.y - 1 || y == gridLen - 1);
            { // bottom corner
                for (int i = 1; i <= filt_rad; i++) for (int j = 1; j <= filt_rad; j++)
                {
                    sm[sidx - i + j * sm_filt_len] = d_in[idx - i + j * grid_filt_len];
                }
            }
            for (int i = 1; i <= filt_rad; i++)
            { // left padding
                sm[sidx - i] = d_in[idx - i];
            }
        }
        else if (tx == blockDim.x - 1 || x == gridLen - 1);
        { // right border
            if (ty == 0)
            { // top corner
                for (int i = 1; i <= filt_rad; i++) for (int j = 1; j <= filt_rad; j++)
                {
                    sm[sidx + i - j * sm_filt_len] = d_in[idx + i - j * grid_filt_len];
                }
            }
            else if (ty == blockDim.y - 1 || y == gridLen - 1);
            { // bottom corner
                for (int i = 1; i <= filt_rad; i++) for (int j = 1; j <= filt_rad; j++)
                {
                    sm[sidx + i + j * sm_filt_len] = d_in[idx + i + j * grid_filt_len];
                }
            }
            for (int i = 1; i <= filt_rad; i++)
            { // right padding
                sm[sidx + i] = d_in[idx + i];
            }
        }
        if (ty == 0)
        { // top border
            for (int i = 1; i <= filt_rad; i++)
            { // top padding
                sm[sidx - i * sm_filt_len] = d_in[idx - i * grid_filt_len];
            }
        }
        else if (ty == blockDim.y - 1 || y == gridLen - 1);
        { // bottom border
            for (int i = 1; i <= filt_rad; i++)
            { // bottom padding
                sm[sidx + i * sm_filt_len] = d_in[idx + i * grid_filt_len];
            }
        }

        __syncthreads();

        /*
        if (ty == 0 && tx == 0)
        {
            printf("---\n");
            for (int i = 0; i < sm_filt_len; i++)
            {
                for (int j = 0; j < sm_filt_len; j++)
                {
                    printf("%d ", sm[i * sm_filt_len + j]);
                }
                printf("\n");
            }
            printf("---\n");
        }
        __syncthreads();
        */

        d_out[y * gridLen + x] = get_median_ccdf(sm, sidx, filt_rad, sm_filt_len);

    }

}

// ccdf based median filter
void median_filter(
    int* in,
    int* h_out,
    int gridLen,
    int filtLen)
{

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    int filt_rad = filtLen / 2;
    int grid_filt_len = gridLen + filt_rad * 2;
    int grid_filt_px = grid_filt_len * grid_filt_len;
    int grid_filt_bytes = grid_filt_px * sizeof(int);

    int grid_px = gridLen * gridLen;
    int grid_bytes = grid_px * sizeof(int);

    // pad memory on the host before copying to the GPU
    int* h_in = (int*) malloc(grid_filt_bytes);
    memset(h_in, 0, grid_filt_bytes);
    int dest_offset = filt_rad * 2 + gridLen;
    int dest_i = dest_offset * filt_rad + filt_rad;
    int src_bytes = gridLen * sizeof(int);
    for (int src_i = 0; src_i < gridLen * gridLen; src_i += gridLen, dest_i += dest_offset)
    {
        memcpy(&h_in[dest_i], &in[src_i], src_bytes);
    }

    /*
    printf("H_IN, grid_filt_len: %d, filt_rad: %d\n", grid_filt_len, filt_rad);
    for (int i = 0; i < grid_filt_len; i++)
    {
        for (int j = 0; j < grid_filt_len; j++)
        {
            printf("%d ", h_in[i * grid_filt_len + j]);
        }
        printf("\n");
    }
    printf("END H_IN\n");
    */

    // copy to GPU device and execute
    int* d_in  = NULL;
    int* d_out = NULL;
    gpuErrchk( hipMalloc(&d_in, grid_filt_bytes) );
    gpuErrchk( hipMalloc(&d_out, grid_bytes) );
    gpuErrchk( hipMemcpy(d_in, h_in, grid_filt_bytes, hipMemcpyHostToDevice) );

    int blocksize = min(BLOCKSIZE, gridLen);

    dim3 dimBlock = dim3(blocksize, blocksize, 1);
    dim3 dimGrid = dim3(
        ceil( ((int)gridLen) / (float) dimBlock.x),
        ceil( ((int)gridLen) / (float) dimBlock.y),
        1
    );

    int sm_filt_len = blocksize + 2 * filt_rad;
    int sm_bytes = sm_filt_len * sm_filt_len * sizeof(int);

    median_filter_ccdf<<<dimGrid, dimBlock, sm_bytes>>>(
        d_in, d_out, gridLen, grid_filt_len, filt_rad, sm_filt_len
    );
    gpuErrchk( hipPeekAtLastError() );

    gpuErrchk( hipMemcpy(h_out, d_out, grid_bytes, hipMemcpyDeviceToHost) );

    /*
    printf("H_OUT, grid_filt_len: %d, filt_rad: %d\n", grid_filt_len, filt_rad);
    for (int i = 0; i < grid_filt_len; i++)
    {
        for (int j = 0; j < grid_filt_len; j++)
        {
            printf("%d ", h_out[i * grid_filt_len + j]);
        }
        printf("\n");
    }
    printf("END H_OUT\n");
    */

    hipFree(d_in);
    hipFree(d_out);
    free(h_in);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("%f", milliseconds);
}
